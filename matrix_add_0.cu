#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

#define N 5

__global__ void matrix_add(int* A, int* B, int* C)
{
	// ������Ҫע�⣬cuda��xy �� �����xy ���෴��
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	C[i * N + j] = A[i * N + j] + B[i * N + j];
}


void print_matrix(int *M)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%d ", M[i * N + j]);
		}
		printf("\n");
	}
}


void check_cuda_error(hipError_t status)
{
	if (status != hipSuccess)
	{
		printf("error: %s\n", hipGetErrorString(status));
		exit(1);
	}
}

int main()
{
	int n_blocks = 1;
	dim3 threads_per_block(N, N);


	int* A = (int*)malloc(N * N * sizeof(int));
	int* B = (int*)malloc(N * N * sizeof(int));
	int* C = (int*)malloc(N * N * sizeof(int));
	

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[i * N + j] = B[i * N + j] = i - j;
		}
	}

	int* cuda_A;
	int* cuda_B;
	int* cuda_C;
	
	hipError_t cuda_status;

	cuda_status = hipMalloc((void**)&cuda_A, N * N * sizeof(int));
	check_cuda_error(cuda_status);

	cuda_status = hipMalloc((void**)&cuda_B, N * N * sizeof(int));
	check_cuda_error(cuda_status);

	cuda_status = hipMalloc((void**)&cuda_C, N * N * sizeof(int));
	check_cuda_error(cuda_status);


	hipMemcpy(cuda_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

	matrix_add << <n_blocks, threads_per_block >> > (cuda_A, cuda_B, cuda_C);

	hipMemcpy(C, cuda_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

	printf("A\n");
	print_matrix(A);

	printf("B\n");
	print_matrix(B);

	printf("C\n");
	print_matrix(C);

	hipFree(cuda_A);
	hipFree(cuda_B);
	hipFree(cuda_C);
	free(A);
	free(B);
	free(C);
	return 0;
}