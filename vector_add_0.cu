#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>


__global__ void vecor_add(int* A, int* B, int* C, int N)
{
	int i = threadIdx.x;

	C[i] = A[i] + B[i];
}

int main()
{
	int N = 8;
	int* A = (int*)malloc(N * sizeof(int));
	int* B = (int*)malloc(N * sizeof(int));
	int* C = (int*)malloc(N * sizeof(int));

	int* cuda_A, *cuda_B, *cuda_C;

	hipMalloc((void**)&cuda_A, N * sizeof(int));
	hipMalloc((void**)&cuda_B, N * sizeof(int));
	hipMalloc((void**)&cuda_C, N * sizeof(int));

	for (int i = 0;i < N;i++)
	{
		A[i] = i - 1;
		B[i] = i + 1;

		//printf("%d %d\n", A[i], B[i]);
	}

	hipError_t cuda_status;

	cuda_status = hipMemcpy(cuda_A, A, N * sizeof(int), hipMemcpyHostToDevice);
	
	if (cuda_status != hipSuccess)
	{
		printf("error: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	cuda_status = hipMemcpy(cuda_B, B, N * sizeof(int), hipMemcpyHostToDevice);

	if (cuda_status != hipSuccess)
	{
		printf("error: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	vecor_add << <1, N >> > (cuda_A, cuda_B, cuda_C, N);

	cuda_status = hipMemcpy(C, cuda_C, N * sizeof(int), hipMemcpyDeviceToHost);
	
	if (cuda_status != hipSuccess)
	{
		printf("error: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	for (int i = 0;i < N;i++)
	{
		printf("%d + %d = %d\n", A[i], B[i], C[i]);
	}

	hipFree(cuda_A);
	hipFree(cuda_B);
	hipFree(cuda_C);

	free(A);
	free(B);
	free(C);

	return 0;
}